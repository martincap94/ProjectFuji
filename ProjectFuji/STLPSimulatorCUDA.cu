#include "hip/hip_runtime.h"
#include "STLPSimulatorCUDA.h"

#include "ShaderManager.h"
#include "STLPUtils.h"
#include "Utils.h"
#include "HeightMap.h"

#include "hip/hip_runtime.h"
#include ""





STLPSimulatorCUDA::STLPSimulatorCUDA(VariableManager * vars, STLPDiagram * stlpDiagram) : vars(vars), stlpDiagram(stlpDiagram) {
	groundHeight = stlpDiagram->P0;
	boxTopHeight = groundHeight + simulationBoxHeight;

	layerVisShader = ShaderManager::getShaderPtr("singleColorAlpha");

	initBuffers();

	
}

STLPSimulatorCUDA::~STLPSimulatorCUDA() {
}

void STLPSimulatorCUDA::initBuffers() {

	glGenVertexArrays(1, &particlesVAO);
	glBindVertexArray(particlesVAO);

	glGenBuffers(1, &particlesVBO);
	glBindBuffer(GL_ARRAY_BUFFER, particlesVBO);

	glEnableVertexAttribArray(0);





	vector<glm::vec3> vertices;

	glGenVertexArrays(1, &CCLLevelVAO);
	glBindVertexArray(CCLLevelVAO);
	glGenBuffers(1, &CCLLevelVBO);
	glBindBuffer(GL_ARRAY_BUFFER, CCLLevelVBO);

	float altitude;
	altitude = getAltitudeFromPressure(stlpDiagram->CCL.y);
	mapToSimulationBox(altitude);
	vertices.push_back(glm::vec3(0.0f, altitude, 0.0f));
	vertices.push_back(glm::vec3(0.0f, altitude, vars->latticeDepth));
	vertices.push_back(glm::vec3(vars->latticeWidth, altitude, vars->latticeDepth));
	vertices.push_back(glm::vec3(vars->latticeWidth, altitude, 0.0f));


	glBufferData(GL_ARRAY_BUFFER, sizeof(glm::vec3) * 4, &vertices[0], GL_STATIC_DRAW);

	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(glm::vec3), (void *)0);

	glBindVertexArray(0);


	vertices.clear();

	glGenVertexArrays(1, &ELLevelVAO);
	glBindVertexArray(ELLevelVAO);
	glGenBuffers(1, &ELLevelVBO);
	glBindBuffer(GL_ARRAY_BUFFER, ELLevelVBO);

	altitude = getAltitudeFromPressure(stlpDiagram->EL.y);
	mapToSimulationBox(altitude);
	vertices.push_back(glm::vec3(0.0f, altitude, 0.0f));
	vertices.push_back(glm::vec3(0.0f, altitude, vars->latticeDepth));
	vertices.push_back(glm::vec3(vars->latticeWidth, altitude, vars->latticeDepth));
	vertices.push_back(glm::vec3(vars->latticeWidth, altitude, 0.0f));

	glBufferData(GL_ARRAY_BUFFER, sizeof(glm::vec3) * 4, &vertices[0], GL_STATIC_DRAW);

	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(glm::vec3), (void *)0);

	glBindVertexArray(0);
}

void STLPSimulatorCUDA::initCUDA() {

	hipMalloc((void**)&d_verticalVelocities, sizeof(float) * maxNumParticles);
	hipMalloc((void**)&d_profileIndices, sizeof(int) * maxNumParticles);
	hipMalloc((void**)&d_particlePressures, sizeof(float) * maxNumParticles);
	
	hipMemset(d_verticalVelocities, 0, sizeof(float) * maxNumParticles);
	hipMemset(d_profileIndices, 0, sizeof(int) * maxNumParticles);
	hipMemset(d_particlePressures, 0, sizeof(float) * maxNumParticles);



}

void STLPSimulatorCUDA::doStep() {
}

void STLPSimulatorCUDA::resetSimulation() {
}

void STLPSimulatorCUDA::generateParticle() {

	float randx = (float)(rand() / (float)(RAND_MAX / ((float)heightMap->width - 2.0f)));
	float randz = (float)(rand() / (float)(RAND_MAX / ((float)heightMap->height - 2.0f)));

	// let's use small square 
	//float randx = (float)(rand() / (float)(RAND_MAX / ((float)GRID_WIDTH / 10.0f - 2.0f)));
	//float randz = (float)(rand() / (float)(RAND_MAX / ((float)GRID_DEPTH / 10.0f - 2.0f)));

	// interpolate
	int leftx = (int)randx;
	int rightx = leftx + 1;
	int leftz = (int)randz;
	int rightz = leftz + 1;

	// leftx and leftz cannot be < 0 and rightx and rightz cannot be >= GRID_WIDTH or GRID_DEPTH
	float xRatio = randx - leftx;
	float zRatio = randz - leftz;

	float y1 = heightMap->data[leftx][leftz];
	float y2 = heightMap->data[leftx][rightz];
	float y3 = heightMap->data[rightx][leftz];
	float y4 = heightMap->data[rightx][rightz];

	float yLeftx = zRatio * y2 + (1.0f - zRatio) * y1;
	float yRightx = zRatio * y4 + (1.0f - zRatio) * y3;

	float y = yRightx * xRatio + (1.0f - xRatio) * yLeftx;

	//rangeToRange(y, 0.0f, GRID_HEIGHT, 0.0f, 15000.0f);
	mapFromSimulationBox(y);
	//cout << y << endl;

	//y = 1500.0f;

	Particle p;
	p.position = glm::vec3(randx, y, randz);
	p.velocity = glm::vec3(0.0f);
	//p.updatePressureVal();
	//p.convectiveTemperature = stlpDiagram->Tc.x;
	p.profileIndex = rand() % (stlpDiagram->numProfiles - 1);
	//p.convectiveTemperature = stlpDiagram->TcProfiles[p.profileIndex].x;

	//cout << "Pressure at " << y << " is " << p.pressure << endl;

	//float tmpP = 943.0f;
	//float tmpz = getAltitudeFromPressure(tmpP);
	//cout << "Altitude at pressure " << tmpP << " is " << tmpz << endl;
	//tmpP = getPressureFromAltitude(tmpz);
	//cout << "Pressure at altitude " << tmpz << " is " << tmpP << endl;

	//tmpP = 100.0f;
	//tmpz = getAltitudeFromPressure(tmpP);
	//cout << "Altitude at pressure " << tmpP << " is " << tmpz << endl;

	//p.position.y = getAltitudeFromPressure(stlpDiagram->soundingData[0].data[PRES]);

	p.updatePressureVal();

	particles.push_back(p);
	particlePositions.push_back(glm::vec3(randx, y, randz));
	numParticles++;


	/*int randx = rand() % (GRID_WIDTH - 1);
	int randz = rand() % (GRID_DEPTH - 1);

	float y = heightMap->data[randx][randz];

	particlePositions.push_back(glm::vec3(randx, y, randz));
	numParticles++;*/
}

void STLPSimulatorCUDA::draw(ShaderProgram & particlesShader) {
	
	//glUseProgram(particlesShader.id);

	//glPointSize(1.0f);
	//particlesShader.setVec4("color", glm::vec4(1.0f, 0.4f, 1.0f, 1.0f));

	//glBindVertexArray(particlesVAO);

	//glBindBuffer(GL_ARRAY_BUFFER, particlesVBO);
	//glBufferData(GL_ARRAY_BUFFER, sizeof(glm::vec3) * numParticles, &particlePositions[0], GL_DYNAMIC_DRAW);
	////glBufferData(GL_ARRAY_BUFFER, sizeof(glm::vec3), &testParticle.position[0], GL_DYNAMIC_DRAW);

	////glDrawArrays(GL_POINTS, 0, numParticles);
	//glDrawArrays(GL_POINTS, 0, numParticles);

	if (showCCLLevelLayer || showELLevelLayer) {
		GLboolean cullFaceEnabled;
		glGetBooleanv(GL_CULL_FACE, &cullFaceEnabled);
		glDisable(GL_CULL_FACE);

		layerVisShader->use();

		if (showCCLLevelLayer) {
			layerVisShader->setVec4("u_Color", glm::vec4(1.0f, 0.0f, 0.0f, 0.2f));

			glBindVertexArray(CCLLevelVAO);
			glDrawArrays(GL_TRIANGLE_FAN, 0, 4);
		}

		if (showELLevelLayer) {
			layerVisShader->setVec4("u_Color", glm::vec4(0.0f, 1.0f, 0.0f, 0.2f));


			glBindVertexArray(ELLevelVAO);
			glDrawArrays(GL_TRIANGLE_FAN, 0, 4);
		}

		if (cullFaceEnabled) {
			glEnable(GL_CULL_FACE);
		}
	}
}

void STLPSimulatorCUDA::initParticles() {
}

void STLPSimulatorCUDA::mapToSimulationBox(float & val) {
	rangeToRange(val, groundHeight, boxTopHeight, 0.0f, vars->latticeHeight);
}

void STLPSimulatorCUDA::mapFromSimulationBox(float & val) {
	rangeToRange(val, 0.0f, vars->latticeHeight, groundHeight, boxTopHeight);
}
