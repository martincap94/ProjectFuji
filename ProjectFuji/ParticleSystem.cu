#include "hip/hip_runtime.h"
#include "ParticleSystem.h"

#include <hip/hip_runtime.h>

#include <iostream>
#include <random>

#include "Utils.h"
#include "LBM.h"
#include "CUDAUtils.cuh"

#include "Emitter.h"
#include "PositionalEmitter.h"
#include "CircleEmitter.h"
//#include "CDFEmitter.h"
#include "CDFEmitter.h"
#include "EmitterBrushMode.h"


#include "TextureManager.h"

#include <thrust\sort.h>
#include <thrust\device_ptr.h>
#include <thrust\execution_policy.h>
#include <thrust\sequence.h>

#include <stdio.h>

#include <filesystem>
namespace fs = std::experimental::filesystem;



__global__ void computeParticleDistances(glm::vec3 *particleVertices, float *particleDistances, glm::vec3 referencePosition, int numParticles) {

	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx < numParticles) {

		particleDistances[idx] = glm::distance(particleVertices[idx], referencePosition);

	}

}

__global__ void computeParticleProjectedDistances(glm::vec3 *particleVertices, float *particleDistances, glm::vec3 sortVector, int numParticles) {

	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx < numParticles) {
		particleDistances[idx] = glm::dot(particleVertices[idx], sortVector);
	}


}


__global__ void checkParticleValidityKernel(glm::vec3 *particleVertices, int numParticles) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx < numParticles) {
		glm::vec3 pos = particleVertices[idx];
		if (isnan(pos.x) || isnan(pos.y) || isnan(pos.z) || isinf(pos.x) || isinf(pos.y) || isinf(pos.z)) {
			//printf("oh no");
			particleVertices[idx] = glm::vec3(0.0f);
		}

		
		/*
		// DO NOT USE THIS - it is much more readable, but it doesn't seem to work correctly on GPU!
		if (glm::any(glm::isnan(pos)) || glm::any(glm::isinf(pos))) {
			particleVertices[idx] = glm::vec3(0.0f);
		}
		*/

	}
}

__global__ void clearVerticalVelocitiesKernel(float *verticalVelocities, int numParticles) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx < numParticles) {
		verticalVelocities[idx] = 0.0f;
	}
}



ParticleSystem::ParticleSystem(VariableManager *vars) : vars(vars) {

	loadParticleSaveFiles();

	curveShader = ShaderManager::getShaderPtr("curve");
	pointSpriteTestShader = ShaderManager::getShaderPtr("pointSpriteTest");
	singleColorShader = ShaderManager::getShaderPtr("singleColor");


	heightMap = vars->heightMap;
	numParticles = vars->numParticles;

	blockDim = dim3(256, 1, 1);
	gridDim = dim3((int)ceil((float)numParticles / (float)blockDim.x), 1, 1);


	numActiveParticles = 0;
	//numActiveParticles = numParticles;

	initBuffers();
	initCUDA();

	//spriteTexture = TextureManager::loadTexture((string)TEXTURES_DIR + "radial-gradient-white-2.png");
	//secondarySpriteTexture = TextureManager::loadTexture((string)TEXTURES_DIR + "radial-gradient-white-2.png");

	spriteTexture = TextureManager::loadTexture((string)TEXTURES_DIR + "testTexture.png");
	secondarySpriteTexture = TextureManager::loadTexture((string)TEXTURES_DIR + "testTexture2.png");


	//spriteTexture.loadTexture(((string)TEXTURES_DIR + "testTexture.png").c_str());
	//secondarySpriteTexture.loadTexture(((string)TEXTURES_DIR + "testTexture2.png").c_str());


	disableAllEmitters();

	//formBoxVisModel = new Model("models/unitbox.fbx");
	formBoxVisShader = ShaderManager::getShaderPtr("singleColorModel");
	formBoxVisModel = new Model("models/unitbox.fbx");



}


ParticleSystem::~ParticleSystem() {
	//delete[] particleVertices;

	CHECK_ERROR(hipGraphicsUnregisterResource(cudaParticleVerticesVBO));
	CHECK_ERROR(hipGraphicsUnregisterResource(cudaParticleProfilesVBO));
	CHECK_ERROR(hipGraphicsUnregisterResource(cudaDiagramParticleVerticesVBO));

	for (int i = 0; i < emitters.size(); i++) {
		delete emitters[i];
	}

	hipFree(d_numParticles);

	if (formBoxVisModel) {
		delete formBoxVisModel;
	}

}


void ParticleSystem::update() {
	if (ebm->isActive()) {
		if (ebm->hasActiveBrush()) {
			ebm->getActiveBrushPtr()->update();
		}
	} else {
		for (int i = 0; i < emitters.size(); i++) {
			emitters[i]->update();
		}
	}
	emitParticles();
}



void ParticleSystem::initBuffers() {

	glGenVertexArrays(1, &particlesVAO);
	glBindVertexArray(particlesVAO);
	glGenBuffers(1, &particleVerticesVBO);
	glBindBuffer(GL_ARRAY_BUFFER, particleVerticesVBO);

	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(glm::vec3), (void *)0);

	glGenBuffers(1, &particleProfilesVBO);
	glBindBuffer(GL_ARRAY_BUFFER, particleProfilesVBO);

	glEnableVertexAttribArray(5);
	glVertexAttribIPointer(5, 1, GL_INT, sizeof(int), (void *)0);

	vector<unsigned int> indices;
	for (int i = 0; i < numParticles; i++) {
		indices.push_back(i);
	}

	glGenBuffers(1, &particlesEBO);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, particlesEBO);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, numParticles * sizeof(unsigned int), indices.data(), GL_STATIC_DRAW);

	glBindVertexArray(0);

	CHECK_ERROR(hipGraphicsGLRegisterBuffer(&cudaParticlesEBO, particlesEBO, cudaGraphicsMapFlagsWriteDiscard));




	///////////////////////////////////////////////////////////////////////////////////////
	// DIAGRAM
	///////////////////////////////////////////////////////////////////////////////////////
	glGenVertexArrays(1, &diagramParticlesVAO);
	glBindVertexArray(diagramParticlesVAO);
	glGenBuffers(1, &diagramParticleVerticesVBO);
	glBindBuffer(GL_ARRAY_BUFFER, diagramParticleVerticesVBO);

	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, sizeof(glm::vec2), (void *)0);

	glBindVertexArray(0);

}


void ParticleSystem::initCUDA() {

	CHECK_ERROR(hipMalloc((void**)&d_numParticles, sizeof(int)));
	CHECK_ERROR(hipMemcpy(d_numParticles, &numParticles, sizeof(int), hipMemcpyHostToDevice));

	CHECK_ERROR(hipMalloc((void**)&d_verticalVelocities, sizeof(float) * numParticles));

	CHECK_ERROR(hipMemset(d_verticalVelocities, 0, sizeof(float) * numParticles));

	//cudaGLRegisterBufferObject(cudaDiagramParticleVerticesVBO, )

	CHECK_ERROR(hipMalloc((void**)&d_particleDistances, sizeof(float) * numParticles));
	CHECK_ERROR(hipMemset(d_particleDistances, 0, sizeof(float) * numParticles));


}

void ParticleSystem::emitParticles() {

	//// check if emitting particles is possible (maximum reached)
	// --> checking is also done in each emitter, this prevents further unnecessary work
	if (numActiveParticles >= numParticles) {
		return;
	}

	int prevNumActiveParticles = numActiveParticles;


	if (ebm->isActive()) {
		if (ebm->hasActiveBrush()) {
			ebm->getActiveBrushPtr()->emitParticles(ebm->numParticlesEmittedPerFrame);
		}
	} else {

		// go through all emitters and emit particles (each pushes them to this system)
		for (int i = 0; i < emitters.size(); i++) {
			emitters[i]->emitParticles();
		}
	}

	//cout << "num particles to upload = " << particleVerticesToEmit.size() << endl;

	// upload the data to VBOs and CUDA memory

	glNamedBufferSubData(particleVerticesVBO, sizeof(glm::vec3) * prevNumActiveParticles, sizeof(glm::vec3) * particleVerticesToEmit.size()/*(numActiveParticles - prevNumActiveParticles)*/, particleVerticesToEmit.data());

	glNamedBufferSubData(particleProfilesVBO, sizeof(int) * prevNumActiveParticles, sizeof(int) * particleProfilesToEmit.size(), particleProfilesToEmit.data());

	//cout << verticalVelocitiesToEmit.size() << endl;
	//cout << " | prevNumActiveParticles = " << prevNumActiveParticles << endl;
	//cout << " | numActiveParticles     = " << numActiveParticles << endl;
	//cout << " | active - prevActive    = " << (numActiveParticles - prevNumActiveParticles) << endl;

	CHECK_ERROR(hipMemcpy(d_verticalVelocities + prevNumActiveParticles, verticalVelocitiesToEmit.data(), verticalVelocitiesToEmit.size() * sizeof(float), hipMemcpyHostToDevice));


	// clear the temporary emitted particle structures

	particleVerticesToEmit.clear();
	particleProfilesToEmit.clear();
	verticalVelocitiesToEmit.clear();

}





// NOT USED ANYMORE
void ParticleSystem::draw(glm::vec3 cameraPos) {


	/*
	size_t num_bytes;
	glm::vec3 *d_mappedParticleVerticesVBO;

	CHECK_ERROR(hipGraphicsMapResources(1, &cudaParticleVerticesVBO, 0));
	CHECK_ERROR(hipGraphicsResourceGetMappedPointer((void **)&d_mappedParticleVerticesVBO, &num_bytes, cudaParticleVerticesVBO));

	CHECK_ERROR(hipGetLastError());

	computeParticleDistances << <gridDim.x, blockDim.x >> > (d_mappedParticleVerticesVBO, d_particleDistances, cameraPos, numActiveParticles);

	CHECK_ERROR(hipGetLastError());

	// this is a no go, we need to sort by key! -> more memory...
	//thrust::device_ptr<glm::vec3> dptr(d_mappedParticleVerticesVBO);  // add this line before the sort line
	//thrust::sort(dptr, dptr + numActiveParticles);        // modify your sort line


	//thrust::sort_by_key(keys.begin(), keys.end(), values.begin());


	//// OLD APPROACH
	//thrust::device_ptr<glm::vec3> thrustParticleVerticesPtr(d_mappedParticleVerticesVBO);
	//thrust::device_ptr<float> thrustParticleDistancesPtr(d_particleDistances);

	//thrust::sort_by_key(thrustParticleDistancesPtr, thrustParticleDistancesPtr + numActiveParticles, thrustParticleVerticesPtr, thrust::greater<float>());


	// NEW APPROACH
	thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::greater<float>());
	CHECK_ERROR(hipGetLastError());


	//hipDeviceSynchronize(); // if we do not synchronize, thrust will (?) throw a system error since we unmap the resource before it finishes sorting
	CHECK_ERROR(hipGraphicsUnmapResources(1, &cudaParticleVerticesVBO, 0));
	*/

	ShaderProgram *shader;
	if (vars->usePointSprites) {

		glEnable(GL_BLEND);
		glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

		glDepthMask(GL_FALSE);
		shader = pointSpriteTestShader;

	} else {
		shader = singleColorShader;
	}

	//glUseProgram(point.id);
	pointSpriteTestShader->use();

	pointSpriteTestShader->setBool("u_ShowHiddenParticles", (bool)showHiddenParticles);
	pointSpriteTestShader->setInt("u_Tex", 0);
	pointSpriteTestShader->setInt("u_SecondTex", 1);
	pointSpriteTestShader->setVec3("u_TintColor", vars->tintColor);

	pointSpriteTestShader->setInt("u_OpacityBlendMode", opacityBlendMode);
	pointSpriteTestShader->setFloat("u_OpacityBlendRange", opacityBlendRange);


	glActiveTexture(GL_TEXTURE0 + 0);
	glBindTexture(GL_TEXTURE_2D, spriteTexture->id);

	glActiveTexture(GL_TEXTURE0 + 1);
	glBindTexture(GL_TEXTURE_2D, secondarySpriteTexture->id);

	glPointSize(pointSize);
	pointSpriteTestShader->setVec3("u_CameraPos", cameraPos);
	pointSpriteTestShader->setFloat("u_PointSizeModifier", pointSize);
	pointSpriteTestShader->setFloat("u_OpacityMultiplier", vars->opacityMultiplier);

	glBindVertexArray(particlesVAO);


	//glDrawArrays(GL_POINTS, 0, numActiveParticles);
	glDrawElements(GL_POINTS, numActiveParticles, GL_UNSIGNED_INT, 0);


	glDepthMask(GL_TRUE);



	for (int i = 0; i < emitters.size(); i++) {
		emitters[i]->draw();
	}



}

void ParticleSystem::drawGeometry(ShaderProgram *shader, glm::vec3 cameraPos) {

	shader->use();

	glPointSize(pointSize);
	shader->setModelMatrix(glm::mat4(1.0));
	shader->setVec3("u_CameraPos", cameraPos);
	shader->setFloat("u_PointSizeModifier", pointSize);
	shader->setBool("u_IsInstanced", false);

	glBindVertexArray(particlesVAO);
	glDrawArrays(GL_POINTS, 0, numActiveParticles);

}



void ParticleSystem::drawDiagramParticles() {
	curveShader->use();
	GLboolean depthTestEnabled;
	glGetBooleanv(GL_DEPTH_TEST, &depthTestEnabled);
	glDisable(GL_DEPTH_TEST);


	glPointSize(2.0f);
	curveShader->setColor(diagramParticlesColor);

	glBindVertexArray(diagramParticlesVAO);
	//glBindBuffer(GL_ARRAY_BUFFER, particlesVBO);
	//glBufferData(GL_ARRAY_BUFFER, sizeof(glm::vec2) * particlePoints.size(), &particlePoints[0], GL_DYNAMIC_DRAW);
	//glNamedBufferData(particlesVBO, sizeof(glm::vec2) * particlePoints.size(), &particlePoints[0], GL_DYNAMIC_DRAW);
	glDrawArrays(GL_POINTS, 0, numDiagramParticlesToDraw);

	if (depthTestEnabled) {
		glEnable(GL_DEPTH_TEST);
	}

}

void ParticleSystem::drawHelperStructures() {
	if (editingFormBox) {
		formBoxVisShader->use();
		formBoxVisShader->setColor(glm::vec3(1.0f, 0.0f, 0.0f));
		formBoxVisModel->transform.position = newFormBoxSettings.position;
		formBoxVisModel->transform.scale = newFormBoxSettings.size;
		formBoxVisModel->update();
		formBoxVisModel->drawWireframe(formBoxVisShader);
	}


	for (int i = 0; i < emitters.size(); i++) {
		emitters[i]->draw();
	}


}


void ParticleSystem::sortParticlesByDistance(glm::vec3 referencePoint, eSortPolicy sortPolicy) {


	size_t num_bytes;
	glm::vec3 *d_mappedParticleVerticesVBO;
	unsigned int *d_mappedParticlesEBO;

	CHECK_ERROR(hipGraphicsMapResources(1, &cudaParticleVerticesVBO, 0));
	CHECK_ERROR(hipGraphicsResourceGetMappedPointer((void **)&d_mappedParticleVerticesVBO, &num_bytes, cudaParticleVerticesVBO));

	CHECK_ERROR(hipGraphicsMapResources(1, &cudaParticlesEBO, 0));
	CHECK_ERROR(hipGraphicsResourceGetMappedPointer((void **)&d_mappedParticlesEBO, &num_bytes, cudaParticlesEBO));


	CHECK_ERROR(hipGetLastError());

	computeParticleDistances << <gridDim.x, blockDim.x >> > (d_mappedParticleVerticesVBO, d_particleDistances, referencePoint, numActiveParticles);

	CHECK_ERROR(hipGetLastError());

	thrust::sequence(thrust::device, d_mappedParticlesEBO, d_mappedParticlesEBO + numActiveParticles);

	switch (sortPolicy) {
		case GREATER:
			thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticlesEBO, thrust::greater<float>());
			//thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::greater<float>());
			break;
		case LESS:
			thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticlesEBO, thrust::less<float>());
			//thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::less<float>());
			break;
		case GEQUAL:
			thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticlesEBO, thrust::greater_equal<float>());
			//thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::greater_equal<float>());
			break;
		case LEQUAL:
			thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticlesEBO, thrust::less_equal<float>());
			//thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::less_equal<float>());
			break;
	}

	CHECK_ERROR(hipGetLastError());


	CHECK_ERROR(hipGraphicsUnmapResources(1, &cudaParticleVerticesVBO, 0));
	CHECK_ERROR(hipGraphicsUnmapResources(1, &cudaParticlesEBO, 0));

	/*
	size_t num_bytes;
	glm::vec3 *d_mappedParticleVerticesVBO;

	CHECK_ERROR(hipGraphicsMapResources(1, &cudaParticleVerticesVBO, 0));
	CHECK_ERROR(hipGraphicsResourceGetMappedPointer((void **)&d_mappedParticleVerticesVBO, &num_bytes, cudaParticleVerticesVBO));

	CHECK_ERROR(hipGetLastError());

	computeParticleDistances << <gridDim.x, blockDim.x >> > (d_mappedParticleVerticesVBO, d_particleDistances, referencePoint, numActiveParticles);

	CHECK_ERROR(hipGetLastError());

	switch (sortPolicy) {
		case GREATER:
			thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::greater<float>());
			break;
		case LESS:
			thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::less<float>());
			break;
		case GEQUAL:
			thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::greater_equal<float>());
			break;
		case LEQUAL:
			thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::less_equal<float>());
			break;
	}

	CHECK_ERROR(hipGetLastError());


	CHECK_ERROR(hipGraphicsUnmapResources(1, &cudaParticleVerticesVBO, 0));

	*/
}

void ParticleSystem::sortParticlesByProjection(glm::vec3 sortVector, eSortPolicy sortPolicy) {


	glm::vec3 *d_mappedParticleVerticesVBO;
	unsigned int *d_mappedParticlesEBO;

	CHECK_ERROR(hipGraphicsMapResources(1, &cudaParticleVerticesVBO, 0));
	CHECK_ERROR(hipGraphicsResourceGetMappedPointer((void **)&d_mappedParticleVerticesVBO, nullptr, cudaParticleVerticesVBO));

	CHECK_ERROR(hipGraphicsMapResources(1, &cudaParticlesEBO, 0));
	CHECK_ERROR(hipGraphicsResourceGetMappedPointer((void **)&d_mappedParticlesEBO, nullptr, cudaParticlesEBO));


	CHECK_ERROR(hipGetLastError());

	computeParticleProjectedDistances << <gridDim.x, blockDim.x >> > (d_mappedParticleVerticesVBO, d_particleDistances, sortVector, numActiveParticles);

	CHECK_ERROR(hipGetLastError());

	thrust::sequence(thrust::device, d_mappedParticlesEBO, d_mappedParticlesEBO + numActiveParticles);



	switch (sortPolicy) {
		case GREATER:
			thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticlesEBO, thrust::greater<float>());
			//thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::greater<float>());
			break;
		case LESS:
			thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticlesEBO, thrust::less<float>());
			//thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::less<float>());
			break;
		case GEQUAL:
			thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticlesEBO, thrust::greater_equal<float>());
			//thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::greater_equal<float>());
			break;
		case LEQUAL:
			thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticlesEBO, thrust::less_equal<float>());
			//thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticleVerticesVBO, thrust::less_equal<float>());
			break;
	}

	//thrust::sort_by_key(thrust::device, d_particleDistances, d_particleDistances + numActiveParticles, d_mappedParticlesEBO);

	CHECK_ERROR(hipGetLastError());


	CHECK_ERROR(hipGraphicsUnmapResources(1, &cudaParticleVerticesVBO, 0));
	CHECK_ERROR(hipGraphicsUnmapResources(1, &cudaParticlesEBO, 0));



}

void ParticleSystem::checkParticleValidity() {


	glm::vec3 *d_mappedParticleVerticesVBO;

	CHECK_ERROR(hipGraphicsMapResources(1, &cudaParticleVerticesVBO, 0));
	CHECK_ERROR(hipGraphicsResourceGetMappedPointer((void **)&d_mappedParticleVerticesVBO, nullptr, cudaParticleVerticesVBO));

	checkParticleValidityKernel << <gridDim.x, blockDim.x >> > (d_mappedParticleVerticesVBO, numActiveParticles);
	
	CHECK_ERROR(hipGetLastError());

	CHECK_ERROR(hipGraphicsUnmapResources(1, &cudaParticleVerticesVBO, 0));

}

void ParticleSystem::initParticlesWithZeros() {
	cout << __FUNCTION__ << " not yet implemented!" << endl;

}

void ParticleSystem::initParticlesOnTerrain() {

	refreshParticlesOnTerrain();

	CHECK_ERROR(hipGraphicsGLRegisterBuffer(&cudaParticleVerticesVBO, particleVerticesVBO, hipGraphicsRegisterFlagsWriteDiscard));
	CHECK_ERROR(hipGraphicsGLRegisterBuffer(&cudaParticleProfilesVBO, particleProfilesVBO, hipGraphicsRegisterFlagsReadOnly)); // this is read only for CUDA!
	CHECK_ERROR(hipGraphicsGLRegisterBuffer(&cudaDiagramParticleVerticesVBO, diagramParticleVerticesVBO, hipGraphicsRegisterFlagsWriteDiscard));

}

void ParticleSystem::initParticlesAboveTerrain() {
	cout << __FUNCTION__ << " not yet implemented!" << endl;
}

void ParticleSystem::formBox() {
	formBox(formBoxSettings.position, formBoxSettings.size);
}

void ParticleSystem::formBox(glm::vec3 pos, glm::vec3 size) {

	vector<glm::vec3> particleVertices;
	for (int i = 0; i < numParticles; i++) {
		particleVertices.push_back(glm::vec3(getRandFloat(pos.x, pos.x + size.x), getRandFloat(pos.y, pos.y + size.y), getRandFloat(pos.z, pos.z + size.z)));
	}



	glNamedBufferData(particleVerticesVBO, sizeof(glm::vec3) * numParticles, particleVertices.data(), GL_STATIC_DRAW);
	clearVerticalVelocities();
}

void ParticleSystem::refreshParticlesOnTerrain() {

	vector<glm::vec3> particleVertices;
	vector<int> particleProfiles;
	vector<float> particlePressures;
	vector<glm::vec2> diagramParticleVertices;

	ppmImage *profileMap = stlpSim->profileMap;
	STLPDiagram *stlpDiagram = stlpSim->stlpDiagram;

	for (int i = 0; i < numParticles; i++) {
		Particle p;
		p.position = heightMap->getRandomWorldPosition();


		glm::ivec3 texelPos = p.position / heightMap->texelWorldSize;

		if (profileMap && profileMap->height >= heightMap->height && profileMap->width >= heightMap->width) {
			p.profileIndex = (rand() % (texelPos.y - texelPos.x) + texelPos.x) % (stlpDiagram->numProfiles - 1);
		} else {
			p.profileIndex = rand() % (stlpDiagram->numProfiles - 1);
		}

		p.updatePressureVal();

		float normP = stlpDiagram->getNormalizedPres(p.pressure);
		glm::vec2 dryAdiabatIntersection = stlpDiagram->dryAdiabatProfiles[p.profileIndex].getIntersectionWithIsobar(normP);
		float particleTemp = stlpDiagram->getDenormalizedTemp(dryAdiabatIntersection.x, normP);

		particleVertices.push_back(p.position);
		particleProfiles.push_back(p.profileIndex);
		diagramParticleVertices.push_back(stlpDiagram->getNormalizedCoords(particleTemp, p.pressure));




		/*

		// LEGACY IMPLEMENTATION


		// testing generation in circle
		float randx;
		float randz;

		int leftx;
		int rightx;
		int leftz;
		int rightz;

		float xRatio;
		float zRatio;

		if (profileMap && profileMap->height >= heightMap->height && profileMap->width >= heightMap->width) {

			float recalculationVal = 0.0f;
			glm::vec3 pif(0.0f);
			int numPositionRecalculations = 0;
			do {
				randx = (float)(rand() / (float)(RAND_MAX / ((float)heightMap->width - 2.0f)));
				randz = (float)(rand() / (float)(RAND_MAX / ((float)heightMap->height - 2.0f)));

				// interpolate
				leftx = (int)randx;
				rightx = leftx + 1;
				leftz = (int)randz;
				rightz = leftz + 1;


				// leftx and leftz cannot be < 0 and rightx and rightz cannot be >= GRID_WIDTH or GRID_DEPTH
				xRatio = randx - leftx;
				zRatio = randz - leftz;

				glm::vec3 p1 = profileMap->data[leftx][leftz];
				glm::vec3 p2 = profileMap->data[leftx][rightz];
				glm::vec3 p3 = profileMap->data[rightx][leftz];
				glm::vec3 p4 = profileMap->data[rightx][rightz];

				glm::vec3 pi1 = zRatio * p2 + (1.0f - zRatio) * p1;
				glm::vec3 pi2 = zRatio * p4 + (1.0f - zRatio) * p3;

				pif = xRatio * pi2 + (1.0f - xRatio) * pi1;
				recalculationVal = pif.z / (float)profileMap->maxIntensity;

				numPositionRecalculations++;

			} while (recalculationVal < positionRecalculationThreshold && numPositionRecalculations < maxPositionRecalculations);

			glm::ivec3 pii = (glm::ivec3)pif;

			if (pii.y != pii.x) {
				p.profileIndex = (rand() % (pii.y - pii.x) + pii.x) % (stlpDiagram->numProfiles - 1);
			} else {
				p.profileIndex = pii.x % (stlpDiagram->numProfiles - 1);
			}

		} else {

			randx = (float)(rand() / (float)(RAND_MAX / ((float)heightMap->width - 2.0f)));
			randz = (float)(rand() / (float)(RAND_MAX / ((float)heightMap->height - 2.0f)));

			// interpolate
			leftx = (int)randx;
			rightx = leftx + 1;
			leftz = (int)randz;
			rightz = leftz + 1;


			// leftx and leftz cannot be < 0 and rightx and rightz cannot be >= GRID_WIDTH or GRID_DEPTH
			xRatio = randx - leftx;
			zRatio = randz - leftz;

			p.profileIndex = rand() % (stlpDiagram->numProfiles - 1);
		}


		float y1 = heightMap->data[leftx + leftz * heightMap->width];
		float y2 = heightMap->data[leftx + rightz * heightMap->width];
		float y3 = heightMap->data[rightx + leftz * heightMap->width];
		float y4 = heightMap->data[rightx + rightz * heightMap->width];

		float yLeftx = zRatio * y2 + (1.0f - zRatio) * y1;
		float yRightx = zRatio * y4 + (1.0f - zRatio) * y3;

		float y = yRightx * xRatio + (1.0f - xRatio) * yLeftx;


		particleVertices.push_back(glm::vec3(randx, y, randz));

		//stlpSim->mapFromSimulationBox(y);

		p.position = glm::vec3(randx, y, randz);
		p.velocity = glm::vec3(0.0f);

		p.updatePressureVal();

		//float particleTemp = stlpDiagram->getDenormalizedTemp(dryAdiabatIntersection.x, normP);

		float normP = stlpDiagram->getNormalizedPres(p.pressure);
		glm::vec2 dryAdiabatIntersection = stlpDiagram->dryAdiabatProfiles[p.profileIndex].getIntersectionWithIsobar(normP);
		float particleTemp = stlpDiagram->getDenormalizedTemp(dryAdiabatIntersection.x, normP);

		diagramParticleVertices.push_back(stlpDiagram->getNormalizedCoords(particleTemp, p.pressure));

		//particles.push_back(p);
		particleProfiles.push_back(p.profileIndex);

		//particlePressures.push_back(p.pressure);
		*/


	}

	glNamedBufferData(particleProfilesVBO, sizeof(int) * particleProfiles.size(), &particleProfiles[0], GL_STATIC_DRAW);


	glNamedBufferData(particleVerticesVBO, sizeof(glm::vec3) * numParticles, particleVertices.data(), GL_STATIC_DRAW);

	glNamedBufferData(diagramParticleVerticesVBO, sizeof(glm::vec2) * numParticles, diagramParticleVertices.data(), GL_STATIC_DRAW);

}

void ParticleSystem::clearVerticalVelocities(bool clearActiveOnly) {
	clearVerticalVelocitiesKernel << <gridDim.x, blockDim.x >> > (d_verticalVelocities, clearActiveOnly ? numActiveParticles : numParticles);
}

void ParticleSystem::activateAllParticles() {
	numActiveParticles = numParticles;
}

void ParticleSystem::deactivateAllParticles() {
	numActiveParticles = 0;
}

void ParticleSystem::activateAllDiagramParticles() {
	numDiagramParticlesToDraw = numActiveParticles;
}

void ParticleSystem::deactivateAllDiagramParticles() {
	numDiagramParticlesToDraw = 0;
}

void ParticleSystem::enableAllEmitters() {
	for (int i = 0; i < emitters.size(); i++) {
		emitters[i]->enabled = true;
	}
}

void ParticleSystem::disableAllEmitters() {
	for (int i = 0; i < emitters.size(); i++) {
		emitters[i]->enabled = false;
	}
}

void ParticleSystem::createPredefinedEmitters() {

	//ech.circleEmitter = new CircleEmitter();
	//ech.cdfEmitter = new CDFEmitter();


	emitters.push_back(new CircleEmitter("Circle test", this, glm::vec3(4000.0f, 0.0f, 4000.0f), 2000.0f));
	emitters.push_back(new CDFEmitter("CDF test", this, "textures/cdf2.png"));
	emitters.push_back(new PositionalCDFEmitter("Positional CDF test", this, "icons/edit.png"));
	emitters.push_back(new CDFEmitter("CDF Dynamic", this, "textures/cdf2.png", true));
	emitters.push_back(new PositionalCDFEmitter("DCGI", this, "textures/dcgi_cdf_mirrored.png"));
	TextureManager::loadTexture("textures/dcgi_cdf.png");
}

void ParticleSystem::createEmitter(int emitterType, string emitterName) {
	Emitter *createdEmitter = nullptr;

	switch (emitterType) {
		case Emitter::eEmitterType::CIRCULAR: {
			createdEmitter = new CircleEmitter(ech.circleEmitter, this);
			break;
		}
		case Emitter::eEmitterType::CDF_TERRAIN: {
			createdEmitter = new CDFEmitter(ech.cdfEmitter, this);
			break;
		}
		case Emitter::eEmitterType::CDF_POSITIONAL: {
			createdEmitter = new PositionalCDFEmitter(ech.pcdfEmitter, this);
			break;
		}
		default:
			break;
	}

	if (createdEmitter != nullptr) {
		createdEmitter->name = emitterName;
		emitters.push_back(createdEmitter);
	}


}

void ParticleSystem::deleteEmitter(int idx) {
	if (idx >= emitters.size()) {
		cout << "Cannot delete emitter at idx " << idx << ": it is out of bounds!" << endl;
		return;
	}
	if (emitters[idx]) {
		delete emitters[idx];
	}
	emitters.erase(emitters.begin() + idx);


}


// Do not use this, it does not work with the CUDA compiler
void ParticleSystem::constructEmitterCreationWindow(nk_context * ctx, UserInterface * ui, int emitterType, bool &closeWindowAfterwards) {
	nk_layout_row_dynamic(ctx, 30, 1);

	static char nameBuffer[64];
	static int nameLength;
	nk_flags event = nk_edit_string(ctx, NK_EDIT_SIMPLE, &nameBuffer[0], &nameLength, 64, nk_filter_default);

	if (event & NK_EDIT_ACTIVATED) {
		vars->generalKeyboardInputEnabled = false;
	}
	if (event & NK_EDIT_DEACTIVATED) {
		vars->generalKeyboardInputEnabled = true;
	}
	nameBuffer[nameLength] = '\0';
	string eName = string(nameBuffer);
	cout << "|" << eName << "|" << endl;
	nk_layout_row_dynamic(ctx, 15, 1);

	switch (emitterType) {
		case Emitter::eEmitterType::CIRCULAR: {
			ech.circleEmitter.constructEmitterPropertiesTab(ctx, ui);
			break;
		}
		case Emitter::eEmitterType::CDF_TERRAIN: {
			ech.cdfEmitter.constructEmitterPropertiesTab(ctx, ui);
			break;
		}
		case Emitter::eEmitterType::CDF_POSITIONAL: {
			ech.pcdfEmitter.constructEmitterPropertiesTab(ctx, ui);
			break;
		}
		default:
			break;

	}

	nk_layout_row_dynamic(ctx, 15, 1);

	if (nk_button_label(ctx, "Create Emitter")) {
		createEmitter(emitterType, eName);
	}
	if (nk_button_label(ctx, "Create and Close")) {
		createEmitter(emitterType, eName);
		closeWindowAfterwards = true;
	}
	if (nk_button_label(ctx, "Close")) {
		closeWindowAfterwards = true;
	}




}

void ParticleSystem::pushParticleToEmit(Particle p) {
	particleVerticesToEmit.push_back(p.position);
	particleProfilesToEmit.push_back(p.profileIndex);
	verticalVelocitiesToEmit.push_back(p.velocity.y);
	numActiveParticles++; // each emitter already checks if numActiveParticles < numParticles, no need to check once more

}

void ParticleSystem::saveParticlesToFile(std::string filename, bool saveOnlyActive) {

	if (!fs::exists(PARTICLE_DATA_DIR)) {
		fs::create_directory(PARTICLE_DATA_DIR);

	} else {
		if (!fs::is_directory(PARTICLE_DATA_DIR)) {
			cout << "Cannot save particles! Please make sure " << PARTICLE_DATA_DIR << " is a directory!" << endl;
			return;
		}
	}

	string fullFilename = PARTICLE_DATA_DIR + filename + ".bin";
	if (fs::exists(fullFilename)) {
		cout << "File " << fullFilename << " exists, will be rewritten!" << endl;
	}



	glm::vec3 *vertexData = (glm::vec3 *)glMapNamedBuffer(particleVerticesVBO, GL_READ_ONLY);
	int *profileData = (int *)glMapNamedBuffer(particleProfilesVBO, GL_READ_ONLY);


	
	ofstream out(fullFilename, ios::binary | ios::out);
	//out << numParticles << endl;
	//out << numActiveParticles << endl;

	int numParticlesToSave = numParticles;
	if (saveOnlyActive) {
		numParticlesToSave = numActiveParticles;
	}

	out.write((char *)&numParticlesToSave, sizeof(int));
	out.write((char *)&numActiveParticles, sizeof(int));

	out.write((char *)&vertexData[0], numParticlesToSave * sizeof(glm::vec3));
	out.write((char *)&profileData[0], numParticlesToSave * sizeof(int));

	//for (int i = 0; i < numParticles; i++) {
	//	out << vertexData[i].x << ' ' << vertexData[i].y << ' ' << vertexData[i].z << ' ' << profileData[i] << endl;
	//}
	//

	glUnmapNamedBuffer(particleVerticesVBO);
	glUnmapNamedBuffer(particleProfilesVBO);



}

void ParticleSystem::constructSaveParticlesWindow(nk_context * ctx, UserInterface * ui, bool & closeWindowAfterwards) {

	static string particleSaveName;
	const static int bufferLength = 32;
	static char nameBuffer[bufferLength];
	static int nameLength;
	static int saveActiveParticlesOnly = 0;

	ui->nk_property_string(ctx, particleSaveName, nameBuffer, bufferLength, nameLength);
	nk_checkbox_label(ctx, "Save Active Particles Only", &saveActiveParticlesOnly);

	if (nameLength == 0) {
		ui->setButtonStyle(ctx, false);
		nk_button_label(ctx, "Save");
		nk_button_label(ctx, "Save and Close");
		ui->setButtonStyle(ctx, true);
	} else {
		if (nk_button_label(ctx, "Save")) {
			saveParticlesToFile(particleSaveName, saveActiveParticlesOnly);
		}
		if (nk_button_label(ctx, "Save and Close")) {
			saveParticlesToFile(particleSaveName, saveActiveParticlesOnly);
			closeWindowAfterwards = true;
		}
	}
	if (nk_button_label(ctx, "Close")) {
		closeWindowAfterwards = true;
	}

}

void ParticleSystem::constructLoadParticlesWindow(nk_context * ctx, UserInterface * ui, bool & closeWindowAfterwards) {
	
	static bool fileSelected = false;
	static string selectedFile;

	nk_layout_row_dynamic(ctx, 15, 1);
	if (nk_combo_begin_label(ctx, fileSelected ? selectedFile.c_str() : "Select file...", nk_vec2(nk_widget_width(ctx), 200))) {
		nk_layout_row_dynamic(ctx, 15, 1);
		if (nk_combo_item_label(ctx, "None", NK_TEXT_LEFT)) {
			fileSelected = false;
			nk_combo_close(ctx);
		}
		for (int i = 0; i < particleSaveFiles.size(); i++) {
			if (nk_combo_item_label(ctx, particleSaveFiles[i].c_str(), NK_TEXT_LEFT)) {
				selectedFile = particleSaveFiles[i];
				fileSelected = true;
				nk_combo_close(ctx);
			}
		}
		nk_combo_end(ctx);
	}

	if (!fileSelected) {
		ui->setButtonStyle(ctx, false);
		nk_button_label(ctx, "Load");
		ui->setButtonStyle(ctx, true);
	} else {
		if (nk_button_label(ctx, "Load")) {
			loadParticlesFromFile(selectedFile);
		}
	}
	if (nk_button_label(ctx, "Close")) {
		closeWindowAfterwards = true;
	}

}

void ParticleSystem::loadParticlesFromFile(std::string filename) {
	
	if (!fs::exists(filename) || !fs::is_regular_file(filename)) {
		cout << "Particle file " << filename << " could not be loaded!" << endl;
		return;
	}

	ifstream infile(filename, ios::binary | ios::in);
	int inNumParticles;
	int inNumActiveParticles;
	//infile >> inNumParticles;
	//infile >> inNumActiveParticles;
	infile.read((char *)&inNumParticles, sizeof(int));
	infile.read((char *)&inNumActiveParticles, sizeof(int));


	if (inNumActiveParticles > inNumParticles) {
		printf("There is something wrong with the loaded file: numParticles (%d) < numActiveParticles (%d)!\n", inNumParticles, inNumActiveParticles);
		return;
	} else if (inNumActiveParticles <= 0 || inNumParticles <= 0) {
		printf("There is something wrong with the loaded file: numParticles (%d) <= 0 || numActiveParticles (%d) <= 0!\n", inNumParticles, inNumActiveParticles);
		return;
	}

	bool bufferSubData = false;
	if (inNumParticles > numParticles) {
		cout << "We do not support loading more particles than the application was configured with!" << endl;
		cout << " | Current configuration = " << numParticles << endl;
		cout << " | Loaded configuration  = " << inNumActiveParticles << endl;
		cout << " | ---> Only " << numParticles << " will be loaded instead..." << endl;
		inNumParticles = numParticles;
	} else if (inNumParticles < numParticles) {
		bufferSubData = true;
	}

	if (inNumActiveParticles > numParticles) {
		inNumActiveParticles = numParticles;
	}
	numActiveParticles = inNumActiveParticles;

	int numToUpload = bufferSubData ? inNumActiveParticles : numParticles;


	/*
	glm::vec3 pPos;
	int pIdx;

	vector<glm::vec3> vertexPositions;
	vector<int> vertexProfiles;

	for (int i = 0; i < inNumParticles; i++) {
		infile >> pPos.x;
		infile >> pPos.y;
		infile >> pPos.z;
		infile >> pIdx;

		vertexPositions.push_back(pPos);
		vertexProfiles.push_back(pIdx);
	}

	if (bufferSubData) {
		glNamedBufferSubData(particleVerticesVBO, 0, sizeof(glm::vec3) * numToUpload, vertexPositions.data());
		glNamedBufferSubData(particleProfilesVBO, 0, sizeof(int) * numToUpload, vertexProfiles.data());
	} else {
		glNamedBufferData(particleVerticesVBO, sizeof(glm::vec3) * numToUpload, vertexPositions.data(), GL_STATIC_DRAW);
		glNamedBufferData(particleProfilesVBO, sizeof(int) * numToUpload, vertexProfiles.data(), GL_STATIC_DRAW);
	}
	*/

	
	glm::vec3 *vertexPositions = new glm::vec3[numToUpload];
	int *vertexProfiles = new int[numToUpload];

	infile.read((char *)&vertexPositions[0], numToUpload * sizeof(glm::vec3));
	infile.read((char *)&vertexProfiles[0], numToUpload * sizeof(int));

	if (bufferSubData) {
		glNamedBufferSubData(particleVerticesVBO, 0, sizeof(glm::vec3) * numToUpload, vertexPositions);
		glNamedBufferSubData(particleProfilesVBO, 0, sizeof(int) * numToUpload, vertexProfiles);
	} else {
		glNamedBufferData(particleVerticesVBO, sizeof(glm::vec3) * numToUpload, vertexPositions, GL_STATIC_DRAW);
		glNamedBufferData(particleProfilesVBO, sizeof(int) * numToUpload, vertexProfiles, GL_STATIC_DRAW);
	}
	delete[] vertexPositions;
	delete[] vertexProfiles;
	


	


}

void ParticleSystem::loadParticleSaveFiles() {
	particleSaveFiles.clear();
	string path = PARTICLE_DATA_DIR;
	string ext = "";
	for (const auto &entry : fs::directory_iterator(path)) {
		if (getFileExtension(entry.path().string(), ext)) {
			if (ext == "bin") {
				particleSaveFiles.push_back(entry.path().string());
			}
		}
	}
	//cout << "Possible Particle Save Files:" << endl;
	//for (int i = 0; i < particleSaveFiles.size(); i++) {
	//	cout << " | " << particleSaveFiles[i] << endl;
	//}

}


//void ParticleSystem::generateParticleOnTerrain(std::vector<glm::vec3>& outVector) {
//
//
//	// testing generation in circle
//	float randx;
//	float randz;
//
//	bool incircle = false;
//	if (incircle) {
//
//		float R = 10.0f;
//
//		static random_device rd;
//		static mt19937 mt(rd());
//		static uniform_real_distribution<float> dist(0.0f, 1.0f);
//
//		float a = dist(mt) * 2.0f * (float)PI;
//		float r = R * sqrtf(dist(mt));
//
//		randx = r * cos(a);
//		randz = r * sin(a);
//
//		randx += heightMap->width / 2;
//		randz += heightMap->height / 2;
//
//	} else {
//		randx = (float)(rand() / (float)(RAND_MAX / ((float)heightMap->width - 2.0f)));
//		randz = (float)(rand() / (float)(RAND_MAX / ((float)heightMap->height - 2.0f)));
//	}
//
//	// interpolate
//	int leftx = (int)randx;
//	int rightx = leftx + 1;
//	int leftz = (int)randz;
//	int rightz = leftz + 1;
//
//	// leftx and leftz cannot be < 0 and rightx and rightz cannot be >= GRID_WIDTH or GRID_DEPTH
//	float xRatio = randx - leftx;
//	float zRatio = randz - leftz;
//
//	float y1 = heightMap->data[leftx][leftz];
//	float y2 = heightMap->data[leftx][rightz];
//	float y3 = heightMap->data[rightx][leftz];
//	float y4 = heightMap->data[rightx][rightz];
//
//	float yLeftx = zRatio * y2 + (1.0f - zRatio) * y1;
//	float yRightx = zRatio * y4 + (1.0f - zRatio) * y3;
//
//	float y = yRightx * xRatio + (1.0f - xRatio) * yLeftx;
//
//	//y = 5.0f; //////////////////////////////////////////////////////// FORCE Y to dry adiabat
//
//	particlePositions.push_back(glm::vec3(randx, y, randz));
//
//
//	mapFromSimulationBox(y);
//
//	Particle p;
//	p.position = glm::vec3(randx, y, randz);
//	p.velocity = glm::vec3(0.0f);
//
//
//	if (profileMap && profileMap->height >= heightMap->height && profileMap->width >= heightMap->width) {
//
//		glm::vec2 p1 = profileMap->data[leftx][leftz];
//		glm::vec2 p2 = profileMap->data[leftx][rightz];
//		glm::vec2 p3 = profileMap->data[rightx][leftz];
//		glm::vec2 p4 = profileMap->data[rightx][rightz];
//
//		glm::vec2 pi1 = zRatio * p2 + (1.0f - zRatio) * p1;
//		glm::vec2 pi2 = zRatio * p4 + (1.0f - zRatio) * p3;
//
//		glm::vec2 pif = xRatio * pi2 + (1.0f - xRatio) * pi1;
//		glm::ivec2 pii = (glm::ivec2)pif;
//
//		if (pii.y != pii.x) {
//			p.profileIndex = (rand() % (pii.y - pii.x) + pii.x) % (stlpDiagram->numProfiles - 1);
//		} else {
//			p.profileIndex = pii.x % (stlpDiagram->numProfiles - 1);
//		}
//
//	} else {
//		p.profileIndex = rand() % (stlpDiagram->numProfiles - 1);
//	}
//
//
//	p.updatePressureVal();
//
//	particles.push_back(p);
//	numParticles++;
//
//}





/*

void ParticleSystem::initParticlePositions(int width, int height, bool *collider) {
	cout << "Initializing particle positions." << endl;
	int particleCount = 0;
	float x = 0.0f;
	float y = 0.0f;
	float offset = 0.5f;
	float xOffset = 0.0f;
	float yOffset = 0.0f;
	while (particleCount != numParticles) {
		if (!collider[(int)x + width * (int)y]) {
			particleVertices[particleCount] = glm::vec3(x, y, -1.0f);
			particleCount++;
		}
		y++;
		if (y >= height - 1) {
			y = yOffset;
			x++;
		}
		if (x >= width - 1) {
			yOffset += offset;
			if (yOffset >= 1.0f) {
				yOffset = 0.0f;
				xOffset += offset;
				if (xOffset >= 1.0f) {
					xOffset = 0.0f;
					offset /= 2.0f;
					yOffset += offset;
				}
			}
			x = xOffset;
			y = yOffset;
		}
	}
	cout << "Particle positions intialized!" << endl;
	glBindBuffer(GL_ARRAY_BUFFER, VBO);

	glBufferData(GL_ARRAY_BUFFER, sizeof(glm::vec3) * numParticles, &particleVertices[0], GL_DYNAMIC_DRAW);
}

void ParticleSystem::initParticlePositions(int width, int height, int depth, const HeightMap *hm) {


	// generate in the left wall
	int particleCount = 0;
	float x = 0.0f;
	float y = 0.0f;
	float z = 0.0f;
	float offset = 0.5f;
	float xOffset = 0.0f;
	float yOffset = 0.0f;
	float zOffset = 0.0f;
	while (particleCount != numParticles) {
		if (hm->data[(int)x][(int)z] <= y) {
			particleVertices[particleCount] = glm::vec3(x, y, z);
			particleCount++;
		}
		z++;
		// prefer depth instead of height
		if (z >= depth - 1) {
			z = zOffset;
			y++;
		}
		if (y >= height - 1) {
			y = yOffset;
			z = zOffset;
			x++;
		}
		if (x >= width - 1) {
			xOffset += offset;
			if (xOffset >= 1.0f) {
				xOffset = 0.0f;
				yOffset += offset;
				if (yOffset > 1.0f) {
					yOffset = 0.0f;
					zOffset += offset;
					if (zOffset >= 1.0f) {
						zOffset = 0.0f;
						offset /= 2.0f;
						xOffset += offset;
					}
				}
			}
			x = xOffset;
			y = yOffset;
			z = zOffset;
		}
	}
	glBindBuffer(GL_ARRAY_BUFFER, VBO);

	glBufferData(GL_ARRAY_BUFFER, sizeof(glm::vec3) * numParticles, &particleVertices[0], GL_DYNAMIC_DRAW);
}

void ParticleSystem::copyDataFromVBOtoCPU() {

	printf("Copying data from VBO to CPU in ParticleSystem\n");
	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glm::vec3 *tmp = (glm::vec3 *)glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);

	for (int i = 0; i < numParticles; i++) {
		particleVertices[i] = tmp[i];
	}
	glUnmapBuffer(GL_ARRAY_BUFFER);


}

*/