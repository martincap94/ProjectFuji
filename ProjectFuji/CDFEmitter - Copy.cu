#include "hip/hip_runtime.h"
#include "CDFEmitter.h"

#include "hip/hip_runtime.h"
#include ""

#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>


#include "CUDAUtils.cuh"
#include "ParticleSystem.h"
#include "TextureManager.h"
#include "Utils.h"

#include <stb_image.h>

using namespace std;

//#define THRUST_BIN_SEARCH // much slower than regular CPU version


// expects path to 16-bit grayscale png
CDFEmitter::CDFEmitter(ParticleSystem *owner, string probabilityTexturePath) : Emitter(owner) {

	//std::uniform_int_distribution<unsigned long long int> idist;

	stbi_set_flip_vertically_on_load(true);
	//stbi_set_flo

	unsigned short *imageData = stbi_load_16(probabilityTexturePath.c_str(), &width, &height, &numChannels, NULL);
	if (!imageData) {
		cout << "Error loading texture at " << probabilityTexturePath << endl;
		stbi_image_free(imageData);
		return;
	}

	sums = new float[width * height]();
	arr = new float[width * height]();
	float *fimgData = new float[width * height]();

	float currSum = 0;
	float maxIntensity = (float)numeric_limits<unsigned short>().max();

	for (int y = 0; y < height; y++) {
		for (int x = 0; x < width; x++) {
			unsigned short *pixel = imageData + (x + y * width) * numChannels;
			unsigned short val = pixel[0];
			currSum += (float)val;
			fimgData[x + y * width] = (float)val / maxIntensity;
			sums[x + y * width] = currSum; // simple sequential inclusive scan (sequential prefix sum)
			arr[x + y * width] = (float)val;
		}
	}
	maxTotalSum = currSum;

	//cout << "Max total sum = " << maxTotalSum << endl;

	firstdist = uniform_real_distribution<float>(1, maxTotalSum);





	GLuint texId;

	glGenTextures(1, &texId);
	glBindTexture(GL_TEXTURE_2D, texId);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_R32F, width, height, 0, GL_RED, GL_FLOAT, fimgData);

	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	TextureManager::pushCustomTexture(texId, width, height, 1, "CDF Emitter test");

	delete[] fimgData;

	CHECK_GL_ERRORS();



	if (imageData) {
		stbi_image_free(imageData);
	}

	initCUDA();

}


CDFEmitter::~CDFEmitter() {
	delete[] sums;
	CHECK_ERROR(hipFree(d_sums));
}

void CDFEmitter::emitParticle() {

	if (!canEmitParticle()) {
		return;
	}

	int selectedRow = height - 1;
	int selectedCol = width - 1;


	int left = 0;
	int right = width * height - 1;

	float randVal = firstdist(mt);

	int idx;

#ifdef THRUST_BIN_SEARCH // much slower than CPU version
	thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(d_sums);
	idx = thrust::distance(d_sums, thrust::lower_bound(thrust::device, d_sums, d_sums + width * height, randVal));
#else

	while (left <= right) {
		idx = (left + right) / 2;
		if (randVal <= sums[idx]) {
			right = idx - 1;
		} else {
			left = idx + 1;
		}
	}
	idx = left;
#endif
	//cout << "idx = " << idx << endl;


	selectedRow = idx / width;
	selectedCol = idx % width;


	Particle p;
	glm::vec3 pos;

	p.profileIndex = rand() % (owner->stlpSim->stlpDiagram->numProfiles - 1);
	p.velocity = glm::vec3(0.0f);

	//cout << pos.x << ", " << pos.y << ", " << pos.z << endl;

	//for (int i = 0; i < 1000; i++) {
		pos = glm::vec3(selectedRow, 0.0f, selectedCol);

		// move inside the texel
		pos.x += getRandFloat(0.0f, 1.0f);
		pos.z += getRandFloat(0.0f, 1.0f);

		pos.x *= owner->heightMap->vars->texelWorldSize; // ugly, cleanup
		pos.z *= owner->heightMap->vars->texelWorldSize; // ugly, cleanup
		pos.y = owner->heightMap->getHeight(pos.x, pos.z, true);
		p.position = pos;


		owner->pushParticleToEmit(p);
	//}

}

void CDFEmitter::update() {
}

void CDFEmitter::draw() {
}

void CDFEmitter::draw(ShaderProgram * shader) {
}

void CDFEmitter::initBuffers() {
}

void CDFEmitter::initCUDA() {

	size_t bsize = sizeof(float) * width * height;
	CHECK_ERROR(hipMalloc((void**)&d_sums, bsize));
	CHECK_ERROR(hipMalloc((void**)&d_arr, bsize));

	/*
	// testing
	float maxIntensity = (float)numeric_limits<unsigned short>().max();
	for (int x = 10; x < 100; x++) {
		for (int y = 10; y < 100; y++) {
			arr[x + y * width] = maxIntensity;
		}
	}
	*/


	CHECK_ERROR(hipMemcpy(d_sums, sums, bsize, hipMemcpyHostToDevice));
	CHECK_ERROR(hipMemcpy(d_arr, arr, bsize, hipMemcpyHostToDevice));


	// now let's test the prefix sum scan from Thrust
	thrust::inclusive_scan(thrust::device, d_arr, d_arr + width * height, d_sums);


	CHECK_ERROR(hipMemcpy(sums, d_sums, bsize, hipMemcpyDeviceToHost));





}
